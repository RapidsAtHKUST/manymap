#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <sys/time.h>
// #include <omp.h>
// #include <hip/hip_cooperative_groups.h>
#include "micro.h"

// using namespace cooperative_groups;

#define MCH 2
#define MIS -4
#define M1 4
#define SCN -1

#define WIDTH 32
#define SHIFT 5

#define NSTREAM 128
// #define FLAG 9

// #define MAXLEN 15000
// #define MAXCIGAR 16384 // 15000
#define MAXMEM 500000 // 15000, 9

__global__ void ksw_extd2_kernel(int qlen, const uint8_t *query, int tlen, const uint8_t *target,
							int8_t q, int8_t e, int8_t q2, int8_t e2, int w, int zdrop, ksw_extz_t *ez) {
	extern __shared__ int8_t smem[];
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int elementskip = blockDim.x * gridDim.x;
	int tlen_ = (tlen + WIDTH - 1) / WIDTH, qlen_ = (qlen + WIDTH - 1) / WIDTH;
	int last_H0_t = 0, H0 = 0;
	int t, r;

	// grid_group g = this_grid();

	int8_t *u, *v, *x, *y, *x2, *y2;
	uint8_t *qr;
	const uint8_t *sf;

	int long_thres = e != e2? (q2 - q) / (e - e2) - 1 : 0;
	if (q2 + e2 + long_thres * e2 > q + e + long_thres * e)
		++long_thres;
	int long_diff = long_thres * (e - e2) - (q2 - q) - e2;

	u = smem;
	v = u + tlen_ * WIDTH, x = v + qlen_ * WIDTH, y = x + qlen_ * WIDTH;
	x2 = y + tlen_ * WIDTH, y2 = x2 + qlen_ * WIDTH, qr = (uint8_t*)(y2 + tlen_ * WIDTH);
	sf = target;

	for (t = tid; t < tlen_ * WIDTH; t += elementskip) {
		u[t] = y[t] = -q - e;
		y2[t] = -q2 - e2;
	}
	for (t = tid; t < qlen_ * WIDTH; t += elementskip) {
		v[t] = x[t] = -q -e;
		x2[t] = -q2 - e2;
	}
	for (t = tid; t < qlen; t += elementskip) qr[t] = query[qlen - 1 - t];

	for (r = 0; r < qlen + tlen - 1; ++r) {
		int st = 0, en = tlen - 1;
		uint8_t *qrr = qr + (qlen - 1 - r);
		int8_t *v8 = v + qlen - r - 1, *x8 = x + qlen - r - 1, *x28 = x2 + qlen - r - 1;

		if (st < r - qlen + 1) st = r - qlen + 1;
		if (en > r) en = r;
		if (st < (r-w+1)>>1) st = (r-w+1)>>1; // take the ceil
		if (en > (r+w)>>1) en = (r+w)>>1; // take the floor

		if (st == 0 && tid == 0) {
			v8[0] = r == 0? -q - e : r < long_thres? -e : r == long_thres? long_diff : -e2;
		}
		if (en >= r && tid == 0) {
			((int8_t*)y)[r] = -q - e, ((int8_t*)y2)[r] = -q2 - e2;
			u[r] = r == 0? -q - e : r < long_thres? -e : r == long_thres? long_diff : -e2;
		}

		// g.sync();
		__syncthreads();

		for (t = st + tid; t <= en; t += elementskip) {
			int8_t z, a, b, a2, b2, vt, ut;
			int8_t st = sf[t], qt = qrr[t];

			z = (st == qt) ? MCH : MIS;
			z = ((st == M1) || (qt == M1)) ? SCN : z;
			ut = u[t];
			
			vt = v8[t];

			a = x8[t] + vt;
			b = y[t] + ut;
			a2 = x28[t] + vt;
			b2 = y2[t] + ut;

			z = MAX(z, MAX(a, MAX(b, MAX(a2, b2))));
			z = MIN(z, MCH);
			u[t] = z - vt;
			v8[t] = z - ut;
			x8[t] = MAX(a - z + q, 0) - q - e;
			y[t] = MAX(b - z + q, 0) - q - e;
			x28[t] = MAX(a2 - z + q2, 0) - q2 - e2;
			y2[t] = MAX(b2 - z + q2, 0) - q2 - e2;
		}

		// g.sync();
		__syncthreads();

		if (tid == 0) {
			if (r > 0) {
				if (last_H0_t >= st && last_H0_t <= en && last_H0_t + 1 >= st && last_H0_t + 1 <= en) {
					int32_t d0 = v8[last_H0_t];
					int32_t d1 = u[last_H0_t + 1];
					if (d0 > d1) H0 += d0;
					else H0 += d1, ++last_H0_t;
				} else if (last_H0_t >= st && last_H0_t <= en) {
					H0 += v8[last_H0_t];
				} else {
					++last_H0_t, H0 += u[last_H0_t];
				}
			} else H0 = v8[0] - q - e, last_H0_t = 0;
			// ksw_apply_zdrop_gpu(ez, 1, H0, r, last_H0_t, zdrop, e2);
			if (r == qlen + tlen - 2 && en == tlen - 1)
				ez->score = H0;
		}
	}
}

void ksw_extd2_gpu(int nkernel, int nblock, int nthread, int qlen, uint8_t *query, int tlen, uint8_t *target, int8_t m, int8_t *mat)
{	
	hipStream_t *streams = (hipStream_t*)malloc(NSTREAM * sizeof(hipStream_t));
	// hipEvent_t startEvent, stopEvent;
	// float ms;
	// CHECK(hipEventCreate(&startEvent));
	// CHECK(hipEventCreate(&stopEvent));

	uint8_t *d_query, *d_target;
	// int8_t *d_mat;
	ksw_extz_t *d_ez;
	// uint32_t *d_cigar;
	// void *d_gm;
	// int *d_last_H0_t, *d_H0;

	ksw_extz_t *ez;
	// uint32_t **h_cigar;
	// int *h_m_cigar;

	CHECK(hipHostMalloc(&ez, nkernel * sizeof(ksw_extz_t)));
	// CHECK(hipHostMalloc((void**)&h_cigar, nkernel * sizeof(uint32_t*)));
	// CHECK(hipHostMalloc((void**)&h_m_cigar, nkernel * sizeof(int)));
	memset(ez, 0, nkernel * sizeof(ksw_extz_t));

	for (int i = 0; i < NSTREAM; ++i) {
		CHECK(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
	}

	CHECK(hipMalloc(&d_query, MAXLEN * sizeof(uint8_t) * NSTREAM)); // max: 12123
	CHECK(hipMalloc(&d_target, MAXLEN * sizeof(uint8_t) * NSTREAM)); // max: 12883
	// CHECK(hipMalloc(&d_mat, m * m * sizeof(int8_t)));
	// CHECK(hipMalloc(&d_mat, m * m * sizeof(int8_t) * NSTREAM));
	CHECK(hipMalloc(&d_ez, sizeof(ksw_extz_t) * nkernel)); // stream -> kernel
	// CHECK(hipMalloc(&d_cigar, MAXCIGAR * sizeof(uint32_t) * nkernel)); // stream -> kernel
	// CHECK(hipMalloc(&d_gm, (size_t)MAXMEM * NSTREAM));
	// CHECK(hipMalloc(&d_last_H0_t, sizeof(int) * NSTREAM));
	// CHECK(hipMalloc(&d_H0, sizeof(int) * NSTREAM));

	// CHECK(hipMemcpy(d_mat, mat, m * m * sizeof(int8_t), hipMemcpyHostToDevice));

	for (int i = 0; i < nkernel; ++i) {
		ksw_reset_extz(&ez[i]);
		// ez[i].cigar = &d_cigar[i * MAXCIGAR];
	}

	double mm_realtime0 = realtime();
/*
	for (int i = 0; i < nkernel; ++i) {
		// CHECK(hipMemcpyAsync(&d_mat[i * m * m], mat, m * m * sizeof(int8_t), hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(&d_query[i * MAXLEN], query, qlen * sizeof(uint8_t), hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(&d_target[i * MAXLEN], target, tlen * sizeof(uint8_t), hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(&d_ez[i], &ez[i], sizeof(ksw_extz_t), hipMemcpyHostToDevice, streams[i])); // TODO: only m_cigar is necessary	
	}
	fprintf(stderr, "[hipMemcpyAsync] Timestamp: %.3f\n", realtime() - mm_realtime0);
*/
	int q = 4, e = 2, q2 = 24, e2 = 1, w = -1, zdrop = 400;
	if (w < 0) w = tlen > qlen? tlen : qlen;
	size_t smem_size = tlen * 8;

	// CHECK(hipEventRecord(startEvent, 0));
/*
	for (int i = 0; i < nkernel; ++i) {
		void *gm = (void*)((int8_t*)d_gm + i * MAXMEM);
		uint8_t *dqi = &d_query[i * MAXLEN];
		uint8_t *dti = &d_target[i * MAXLEN];
		ksw_extz_t *dezi = &d_ez[i];

		void *args[] = {&gm, &qlen, &dqi, &tlen, &dti, &q, &e, &q2, &e2, &w, &zdrop, &dezi};
		hipLaunchKernel((void*)ksw_extd2_kernel, nblock, nthread, args, 0, streams[i]);
		// CHECK(hipDeviceSynchronize());
	}
	fprintf(stderr, "[kernel] Timestamp: %.3f\n", realtime() - mm_realtime0);
*/
	for (int i = 0; i < nkernel; ++i) {
		int sid = i % NSTREAM;
		CHECK(hipMemcpyAsync(&d_query[sid * MAXLEN], query, qlen * sizeof(uint8_t), hipMemcpyHostToDevice, streams[sid]));
		CHECK(hipMemcpyAsync(&d_target[sid * MAXLEN], target, tlen * sizeof(uint8_t), hipMemcpyHostToDevice, streams[sid]));
		CHECK(hipMemcpyAsync(&d_ez[i], &ez[i], sizeof(ksw_extz_t), hipMemcpyHostToDevice, streams[sid]));
		ksw_extd2_kernel<<<nblock, nthread, smem_size, streams[sid]>>>(qlen, &d_query[sid * MAXLEN], tlen, &d_target[sid * MAXLEN], q, e, q2, e2, w, zdrop, &d_ez[i]);
		CHECK(hipMemcpyAsync(&ez[i], &d_ez[i], sizeof(ksw_extz_t), hipMemcpyDeviceToHost, streams[sid]));
	}
/*
	for (int i = 0; i < nkernel; ++i) {
		CHECK(hipMemcpyAsync(&ez[i], &d_ez[i], sizeof(ksw_extz_t), hipMemcpyDeviceToHost, streams[i]));
	}
	fprintf(stderr, "[hipMemcpyAsync] Timestamp: %.3f\n", realtime() - mm_realtime0);
*/
	CHECK(hipDeviceSynchronize());
	// CHECK(hipEventRecord(stopEvent, 0));
	// CHECK(hipEventSynchronize(stopEvent));
	// CHECK(hipEventElapsedTime(&ms, startEvent, stopEvent));

	// fprintf(stderr, "[hipDeviceSynchronize] Timestamp: %.3f\n", realtime() - mm_realtime0);
/*
	for (int i = 0; i < nkernel; ++i) {
		printf("max: %u, zdropped: %u, max_q: %d, max_t: %d, mqe: %d, mqe_t: %d, mte: %d, mte_q: %d, score: %d, m_cigar: %d, n_cigar: %d, reach_end: %d\n",
			ez[i].max, ez[i].zdropped, ez[i].max_q, ez[i].max_t, ez[i].mqe, ez[i].mqe_t, ez[i].mte, ez[i].mte_q, ez[i].score, ez[i].m_cigar, ez[i].n_cigar, ez[i].reach_end);
	}
*/
	double curtime = realtime();
	// fprintf(stderr, "[M::%s] Real time: %.3f sec; CPU: %.3f sec; Peak RSS: %.3f GB\n", __func__, realtime() - mm_realtime0, cputime(), peakrss() / 1024.0 / 1024.0 / 1024.0);
	fprintf(stderr, "%.3f GCUPS, %.3f seconds, %.3f GB\n", (double)nkernel * qlen * tlen / (curtime - mm_realtime0) / 1000000000, curtime - mm_realtime0, peakrss() / 1024.0 / 1024.0 / 1024.0);
	// CHECK(hipDeviceReset());
}

int main(int argc, char **argv) {
	int nkernel = atoi(argv[1]);
	int nblock = atoi(argv[2]);
	int nthread = atoi(argv[3]);

	FILE * fp;
	fp = fopen(argv[4], "r");

	int qlen, tlen;
	uint8_t *query, *target;
	CHECK(hipHostMalloc(&query, MAXLEN * sizeof(uint8_t)));
	CHECK(hipHostMalloc(&target, MAXLEN * sizeof(uint8_t)));
	char tmps[128];

	fscanf(fp, "%s%s", tmps, query);
	fscanf(fp, "%s%s", tmps, target);
	qlen = strlen((char*)query);
	tlen = strlen((char*)target);
	for (int i = 0; i < qlen; ++i) {
		query[i] = seq_nt4_table[query[i]];
	}
	for (int i = 0; i < tlen; ++i) {
		target[i] = seq_nt4_table[target[i]];
	}
	fclose(fp);

	int8_t *mat;
	CHECK(hipHostMalloc(&mat, 25 * sizeof(int8_t)));
	ksw_gen_simple_mat(5, mat, 2, 4, 1);

	// CHECK(hipDeviceReset());
	ksw_extd2_gpu(nkernel, nblock, nthread, qlen, query, tlen, target, 5, mat);
	// CHECK(hipDeviceSynchronize());

	return 0;
}
